#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>

#include <stdio.h>
#include "util.h"

#include <omp.h>

#include "testCuda.h"

void testDot();
__global__ void functionCuda(float * x, float * y, int * k);

int main()
{

	printf("hello world");
	testDot();

	return 1;
}



void testDot()
{
	hipError_t cudaStatus;

	int k = 4;

	float * x = (float*)malloc(sizeof(float)* k);
	float * y = (float*)malloc(sizeof(float)* k);

	x[0] = 1;
	x[1] = 2;
	x[2] = 3;
	x[3] = 4;


	y[0] = 1;
	y[1] = 2;
	y[2] = 3;
	y[3] = 4;


	float * dev_x;
	float * dev_y;
	int * dev_k;


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	MyCudaMalloc((void**)&dev_k, sizeof(int), 8);


	MyCudaMalloc((void**)&dev_x, sizeof(float)* k, 1);

	MyCudaMalloc((void**)&dev_y, sizeof(float)* k, 2);

	MyCudaCopy(dev_x, x, sizeof(float)*k, hipMemcpyHostToDevice, 3);
	MyCudaCopy(dev_y, y, sizeof(float)*k, hipMemcpyHostToDevice, 4);
	MyCudaCopy(dev_k, &k, sizeof(int), hipMemcpyHostToDevice, 5);


	functionCuda <<< 1,1 >> >(dev_x, dev_y, dev_k);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test dot launch failed: %s\n", hipGetErrorString(cudaStatus));

	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching loopOverPoints2!\n", cudaStatus);

	}


	MyCudaFree(dev_k);
	MyCudaFree(dev_x);
	MyCudaFree(dev_y);
	


}


__global__ void functionCuda(float * x, float * y, int * k)
{

	double sum = dot(x, y, k);

	printf("sum = %lf ", sum);
}