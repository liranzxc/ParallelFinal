#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>

#include <stdio.h>
#include "util.h"
#include <math.h>
#include <omp.h>
#include <stdarg.h>


__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}


void MyCudaMalloc(void** dev_pointer, size_t size, int error_label)
{
	hipError_t cudaStatus;

	// points malloc n dims  .
	cudaStatus = hipMalloc(dev_pointer, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! error_label : %d ", error_label);

		MyCudaFree(*dev_pointer);
	}


}

void MyCudaCopy(void* dest, void * src, size_t size, hipMemcpyKind kind, int error_label)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(dest, src, size, kind);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! error_label : %d", error_label);
	}


}

void MyCudaFree(void * object)
{
	hipFree(object);
}

__device__ float dot(float * dev_w, float * dev_x,int indexValues, int * dev_k)
{

	float sum = 0;
	for (int i = 0; i < *dev_k + 1; i++)
	{
		sum += dev_w[i] * dev_x[i+ indexValues];

	}

	return sum;

}

//(dev_alfa, dev_values, dev_index_values, dev_W);
__global__ void createNewWeight(float * dev_alfa, float *dev_values,int * indexerValues, float * W_dev)
{
	int i = threadIdx.x;
	W_dev[i] = (*dev_alfa)*dev_values[*indexerValues + i] + W_dev[i];

}



__global__ void	getMisLeadArrayFromPoints(Point * dev_pts, float* dev_values ,float * dev_W, int * dev_mislead, int * dev_k,int * dev_n) {


	int i = blockIdx.x * 1000 + threadIdx.x;

	if (i < *dev_n)
	{
		
	//	printf("working on point[%d] = group = %d \n ", i,dev_pts[i].group);

		int indexValues = i *(*dev_k + 1);

	//	printf("values of points[%d] are => (%f,%f,%f,%f) \n", i, dev_values[indexValues],
		//	dev_values[indexValues+1], dev_values[indexValues+2], dev_values[indexValues + 3]);


		// calaculate fx 

		float fx = dot(dev_W, dev_values,indexValues, dev_k);

//		printf("dot for point[%d] = %f \n", i, fx);

		int sign = fx >= 0 ? 1 : -1;

		if (dev_pts[i].group != sign)   // A group ,mislead
		{
			sign = (dev_pts[i].group - sign) / 2;

			dev_mislead[i] = sign;

		}
		else
		{
			dev_mislead[i] = 0;

		}


	}
}

double ProcessAlfa(Point * dev_pts,float* dev_values, float  * alfa, int *dev_n
	, int *dev_k, int limit, float QC, int n, int k,float ** WSaved)
{
	*WSaved = (float*)malloc((k + 1) * sizeof(float)); // W k+1 dims 
	int * tempresult = (int*)malloc(n * sizeof(int)); // 
	int * mislead = (int*)malloc(n * sizeof(int)); // array of n points , mislead points will be 1 or -1 ,currect=0


	int * dev_mislead = NULL;
	float * dev_W = NULL;
	float * dev_alfa = NULL;
	int * dev_tempresult = NULL;
	hipError_t cudaStatus;

#pragma region malloc and copy values to GPU


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);

		return -2;
	}

	// w , mislead_pts , dev_alfa,tempresult,


	MyCudaMalloc((void**)&dev_W, sizeof(float)* (k + 1), 7);
	hipMemset(dev_W, 0, sizeof(float)* (k + 1));

	//MyCudaCopy(dev_W, W, sizeof(float)*(k + 1), hipMemcpyHostToDevice, 8);

	MyCudaMalloc((void**)&dev_mislead, sizeof(int)* (n), 9);
	hipMemset(dev_mislead, 0, sizeof(int)* (n));

	//MyCudaCopy(dev_mislead, mislead, sizeof(int)*(n), hipMemcpyHostToDevice, 10);

	MyCudaMalloc((void**)&dev_alfa, sizeof(float), 11);
	MyCudaCopy(dev_alfa, alfa, sizeof(float), hipMemcpyHostToDevice, 12);

	MyCudaMalloc((void**)&dev_tempresult, sizeof(int)*n, 13);
	MyCudaCopy(dev_tempresult, tempresult, sizeof(int)*n, hipMemcpyHostToDevice, 14);
	hipMemset(dev_tempresult, 0, sizeof(int)* (n));

	MyCudaCopy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice, 14);




	//// TODO get all values from devices;
	//Point * pts2 = (Point*)malloc(sizeof(Point)*n);


	//MyCudaCopy(W,dev_W, sizeof(float)*(k + 1), hipMemcpyDeviceToHost, 100);
	//MyCudaCopy(mislead, dev_mislead, sizeof(int)*(n), hipMemcpyDeviceToHost, 101);
	//MyCudaCopy(alfa, dev_alfa, sizeof(float), hipMemcpyDeviceToHost, 120);
	//MyCudaCopy(tempresult, dev_tempresult, sizeof(int)*n, hipMemcpyDeviceToHost, 140);

//MyCudaCopy(pts2, dev_pts, sizeof(int)*n, hipMemcpyDeviceToHost, 145);

	//MyCudaCopy(&k, dev_k, sizeof(int), hipMemcpyDeviceToHost, 14);
	//
	//	printf("i=%d0 , cuda w : %lf \n",0, W[0]);
	//	printf(" cuda alfa : %lf \n", *alfa);
	//	printf("i=%d,cuda mislead : %d \n",0, mislead[0]);
	//	printf("i=%d,cuda tempresult : %d \n",0, tempresult[0]); // WORKS
	//printf("cuda n = %d  \n", n);
	//

	//printf("point 0  = (%f,%f,%f,%f)  group = %d \n", pts2[3].values[0], pts2[3].values[1], pts2[3].values[2] ,pts2[3].values[3],pts2[3].group);




#pragma  endregion


	//printf("start computing  Process alfa %f \n",*alfa);
	int threadDims = 1000;
	int blockDims = (n / threadDims) + 1;
	int counter_limit = 0;

	while (counter_limit < limit)
	{

		getMisLeadArrayFromPoints << <blockDims, threadDims >> > (dev_pts, dev_values, dev_W, dev_mislead, dev_k, dev_n);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "loopOverPoints2 launch failed: %s\n", hipGetErrorString(cudaStatus));
			FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
			return -2;

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching loopOverPoints! : %s \n", cudaStatus, hipGetErrorString(cudaStatus));
			FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
			return -2;

		}

		MyCudaCopy(mislead, dev_mislead, (n) * sizeof(int), hipMemcpyDeviceToHost, 15);


		int indexerMiss = 0;
		int result;
		// we get a array of mislead values (0 == ok , else (1,-1) false));
		for (indexerMiss = 0; indexerMiss < n; indexerMiss++)
		{
			result = mislead[indexerMiss];
			if (result == -1 || result == 1) // found point that mislead
				break;
		}

		if (indexerMiss == n)
		{
			// all point in good places
			break;
		}
		else
		{
			// need to create a new W 

			*alfa = *alfa*mislead[indexerMiss]; // alfa * sign
			MyCudaCopy(dev_alfa, alfa, sizeof(float), hipMemcpyHostToDevice, 77);

			*alfa = fabs(*alfa); // back to postive alfa
			
			int indexValues = indexerMiss * (k + 1);
			int * dev_index_values = NULL;
			MyCudaMalloc((void**)&dev_index_values, sizeof(int), 88);
			MyCudaCopy(dev_index_values, &indexValues, sizeof(int), hipMemcpyHostToDevice, 99);
			createNewWeight << <1, k + 1 >> > (dev_alfa, dev_values, dev_index_values, dev_W);

			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "createNewWeight launch failed: %s\n", hipGetErrorString(cudaStatus));
				FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
				return -2;

			}
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching createNewWeight! : %s \n", cudaStatus, hipGetErrorString(cudaStatus));
				FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
				return -2;

			}


			//MyCudaCopy(W, dev_W, sizeof(float)*(k + 1), hipMemcpyDeviceToHost, 70);

			//printf("new W = [%f ,%f,%f,%f] \n", (W)[0], (W)[1], (W)[2], (W)[3]);


		}


		counter_limit++;
	}

	// need to calcate the q , get all mislead point 



	getMisLeadArrayFromPoints << <blockDims, threadDims >> > (dev_pts, dev_values, dev_W, dev_tempresult, dev_k, dev_n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getMisLeadArrayFromPoints Second Time launch failed: %s\n", hipGetErrorString(cudaStatus));
		FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
		return -2;

	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching getMisLeadArrayFromPoints Second Time! : %s \n", cudaStatus, hipGetErrorString(cudaStatus));
		FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
		return -2;

	}

	MyCudaCopy(tempresult, dev_tempresult, (n) * sizeof(int), hipMemcpyDeviceToHost, 15);

	int sumOFmisLead = 0;
	//#pragma omp parallel for reduction(+:sumOFmisLead)
		for (int i = 0; i < n; i++)
		{
			if (tempresult[i] != 0)
			{
				sumOFmisLead += 1;
			}
		}

		//hipMemset(dev_W, 0, sizeof(float)* (k + 1)); // clean up
		//hipMemset(dev_mislead, 0, sizeof(int)* (n));
		//hipMemset(dev_tempresult, 0, sizeof(int)* (n));


		double q = sumOFmisLead / (n*(1.0));

		MyCudaCopy(*WSaved, dev_W, sizeof(float)*(k + 1), hipMemcpyDeviceToHost, 70); // copy W


		FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);

		if (q <= QC)
				return q;
		else
			return 2; // q that never will get and larger from all q possiblies .



}


hipError_t FreeFunction(float * dev_W ,float * dev_alfa, int * dev_mislead ,int * dev_tempresult)
{
	hipError_t cudaStatus;


		cudaStatus = hipFree(dev_W);
		if (cudaStatus != hipSuccess) {

			printf("failed to free cuda - W  \n");
		}
		cudaStatus = hipFree(dev_mislead);
		if (cudaStatus != hipSuccess) {

			printf("failed to free cuda - mislead points \n");
		}
	
		cudaStatus = hipFree(dev_tempresult);
		if (cudaStatus != hipSuccess) {
			
			printf("failed to free cuda - tempresult \n");
		}

		cudaStatus = hipFree(dev_alfa);
		if (cudaStatus != hipSuccess) {

			printf("failed to free cuda - alfa \n");
		}
	
	return cudaStatus;
}

